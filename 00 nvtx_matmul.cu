#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <nvtx3/nvToolsExt.h>
#include <iostream>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define BLOCK_SIZE 16

__global__ void matrixMulKernel(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < N && col < N) {
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void matrixMul(float* A, float* B, float* C, int N) {
    nvtxRangePush("Matrix Multiplication");

    float* d_A, * d_B, * d_C;
    int size = N * N * sizeof(float);

    nvtxRangePush("Memory Allocation");
    hipError_t err;
    err = hipMalloc(&d_A, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for d_A: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_B, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for d_B: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMalloc(&d_C, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc error for d_C: " << hipGetErrorString(err) << std::endl;
        return;
    }
    nvtxRangePop();

    nvtxRangePush("Memory Copy H2D");
    err = hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error from A to d_A: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error from B to d_B: " << hipGetErrorString(err) << std::endl;
        return;
    }
    nvtxRangePop();

    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);

    nvtxRangePush("Kernel Execution");
    matrixMulKernel << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C, N);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "CUDA device synchronization error: " << hipGetErrorString(err) << std::endl;
        return;
    }
    nvtxRangePop();

    nvtxRangePush("Memory Copy D2H");
    err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "CUDA memcpy error from d_C to C: " << hipGetErrorString(err) << std::endl;
        return;
    }
    nvtxRangePop();

    nvtxRangePush("Memory Deallocation");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    nvtxRangePop();

    nvtxRangePop();  // End of Matrix Multiplication
}

int main() {
    const int N = 1024;
    float* A = new float[N * N];
    float* B = new float[N * N];
    float* C = new float[N * N];

    // Initialize matrices A and B here...
    // Example initialization
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
    }

    matrixMul(A, B, C, N);

    // Use result in C...

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}


